#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>

//#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


__global__
void add(unsigned int N, float *a, float *b)
{
    auto index = blockIdx.x * blockDim.x + threadIdx.x;
    auto stride = blockDim.x * gridDim.x;

    for (auto i = index; i < N; i += stride)
        a[i] += b[i];
}

//__global__
//void add2(thrust::device_vector<float> &a, thrust::device_vector<float> &b)
//{
//    thrust::transform(a.begin(), a.end(), b.begin(), b.begin(), [] (auto a, auto b)
//    {
//        return a + b;
//    });
//}

void wrapper()
{
    auto constexpr N = 1'048'576u;

    auto constexpr kBLOCK_DIM = 256;
    auto constexpr kGRID_DIM = (N + kBLOCK_DIM - 1) / kBLOCK_DIM;

    /*thrust::device_vector<float> _a(N, 1.f);
    thrust::device_vector<float> _b(N, 2.f);

    add2(_a, _b);*/

    float *a, *b;

    hipMallocManaged(&a, N * sizeof(float));
    hipMallocManaged(&b, N * sizeof(float));

    for (auto i = 0u; i < N; ++i) {
        a[i] = 1.f;
        b[i] = 2.f;
    }

    add<<<kGRID_DIM, kBLOCK_DIM>>>(N, a, b);

    hipDeviceSynchronize();

    auto maxError = 0.f;

    for (auto i = 0u; i < N; ++i)
        maxError = std::fmax(maxError, std::fabs(a[i] - 3.f));

    std::cout << "Max error: " << maxError << std::endl;

    hipFree(b);
    hipFree(a);
}
